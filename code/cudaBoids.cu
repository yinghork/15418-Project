#include "hip/hip_runtime.h"
#include <algorithm>
#include <string>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaBoids.h"

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalDataConstants {
    int imageWidth;
    int imageHeight;

    int boidCount;
    boid_t *inData;
    boid_t *outData;
};

struct GlobalFlockConstants {
    float driveFactor;
    float maxSpeed;
    float squareMaxSpeed;
    float squareNeighborRadius;
    float squareAvoidanceRadius;

    float cohesionWeight;
    float alignmentWeight;
    float separationWeight;
    float centeringWeight;
};

__constant__ GlobalDataConstants cuDataParams;
__constant__ GlobalFlockConstants cuFlockParams;

////////////////////////////////////////////////////////////////////////////////////////

__global__ void copyFrame() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= cuDataParams.boidCount)
        return;

    cuDataParams.inData[i] = cuDataParams.outData[i];
}

// https://github.com/NVIDIA/cuda-samples/blob/master/Common/hip/hip_vector_types.h
inline __host__ __device__ float2 operator-(float2 &a) {
    return make_float2(-a.x, -a.y);
}

inline __host__ __device__ float2 operator+(float2 a, float2 b) {
    return make_float2(a.x + b.x, a.y + b.y);
}

inline __host__ __device__ void operator+=(float2 &a, float2 b) {
    a.x += b.x;
    a.y += b.y;
}

inline __host__ __device__ float2 operator-(float2 a, float2 b) {
    return make_float2(a.x - b.x, a.y - b.y);
}

inline __host__ __device__ void operator-=(float2 &a, float2 b) {
    a.x -= b.x;
    a.y -= b.y;
}

inline __host__ __device__ float2 operator*(float2 a, float b) {
    return make_float2(a.x * b, a.y * b);
}

inline __host__ __device__ void operator*=(float2 &a, float b) {
    a.x *= b;
    a.y *= b;
}

inline __host__ __device__ float2 operator/(float2 a, float b) {
    return make_float2(a.x / b, a.y / b);
}

inline __host__ __device__ void operator/=(float2 &a, float b) {
    a.x /= b;
    a.y /= b;
}


// My helper functions
__device__ __inline__ float2 toFloat2(pos_t pos) {
    return make_float2(pos.x, pos.y);
}

__device__ __inline__ float2 toFloat2(vel_t vel) {
    return make_float2(vel.x, vel.y);
}

__device__ __inline__ pos_t toPos(float2 f) {
    pos_t pos;
    pos.x = f.x;
    pos.y = f.y;
    return pos;
}

__device__ __inline__ vel_t toVel(float2 f) {
    vel_t vel;
    vel.x = f.x;
    vel.y = f.y;
    return vel;
}

// Float math helper functions
__device__ __inline__ float sqrMagnitude(float2 f) {
    return f.x * f.x + f.y * f.y;
}

__device__ __inline__ float sqrDist(float2 f1, float2 f2) {
    float distX = f1.x - f2.x;
    float distY = f1.y - f2.y;
    return distX * distX + distY * distY;
}

__device__ __inline__ float2 normalize(float2 f) {
    float magnitude = sqrt(f.x * f.x + f.y * f.y);
    return f / magnitude;
}

// Calculate all the steers to apply to the boid
__device__ __inline__ float2 calculate_move(int i) {
    int boidCount = cuDataParams.boidCount;
    
    boid_t *inData = cuDataParams.inData;
    float2 pos = toFloat2(inData[i].position);
    float2 vel = toFloat2(inData[i].velocity);
   
    float2 move = make_float2(0.f, 0.f);
    
    // Extra force (indep of neighbors) to stay on screen
    float2 ceMove = make_float2(0.f, 0.f);

    float2 centerOffset = -pos;
    float t = sqrMagnitude(centerOffset) / cuDataParams.imageWidth;

    // t = 0 when at center, 1 if at edge, so start applying when 90% to edge
    if (t > 0.81f) {
        ceMove *= t * t;
    }

    float c = cuFlockParams.centeringWeight;
    if (sqrMagnitude(ceMove) > c * c) {
        ceMove = normalize(ceMove);
        ceMove *= c;
    }
 
    float2 coMove = make_float2(0.f, 0.f);
    float2 alMove = make_float2(0.f, 0.f);
    float2 seMove = make_float2(0.f, 0.f);

    int neighborCount = 0;
    int separateCount = 0;

    /* Calculate each individual force */
    for (int j = 0; j < boidCount; j++) {
        if (i != j) {
            boid_t otherBoid = inData[j];
            float2 otherPos = toFloat2(otherBoid.position);
            
            float dist = sqrDist(pos, otherPos);
            if (dist < cuFlockParams.squareNeighborRadius) {
                neighborCount++;

                // (Coherence) accumulate positions
                coMove += otherPos;
                
                // (Alignment) accumulate velocities
                alMove += toFloat2(otherBoid.velocity);
                
                if (dist < cuFlockParams.squareAvoidanceRadius) {
                    separateCount++;

                    // (Separation) accumulate position offset
                    seMove += (pos - otherPos);
                }
            }
        }
    }
    
    // If there are no neighbors, maintain velocity & apply centering
    if (neighborCount == 0) {
        move = vel + ceMove;
        return move;
    }
    
    // Average the forces
    if (neighborCount > 0) {
        coMove /= neighborCount;
        alMove /= neighborCount;
    }
    if (separateCount > 0) {
        seMove /= separateCount;
    }

    // Convert cohesion to an offset from current position
    coMove -= pos;
    // Convert alignment to offset from current velocity
    alMove -= vel;

    /* Weigh and combine the forces */
    float k = cuFlockParams.cohesionWeight;
    float m = cuFlockParams.alignmentWeight;
    float s = cuFlockParams.separationWeight;

    coMove *= k;
    alMove *= m;
    seMove *= s;

    // Start combining forces
    if (sqrMagnitude(coMove) > k * k) {
        coMove = normalize(coMove);
        coMove *= k;
    }
    if (sqrMagnitude(alMove) > m * m) {
        alMove = normalize(alMove);
        alMove *= m;
    }
    if (sqrMagnitude(seMove) > s * s) {
        seMove = normalize(seMove);
        seMove *= s;
    }

    move += coMove + alMove + seMove + ceMove;
   
    /* 
    if (i != 0)
        return move;

    printf("Boid %d has move %lf %lf: cohere %lf %lf, align %lf %lf, avoid %lf %lf, center %lf %lf.\n",
            i, move.x, move.y, coMove.x, coMove.y, alMove.x, alMove.y, seMove.x, seMove.y, ceMove.x, ceMove.y);
    printf("Boid %d had %d boids in neighborhood of %lf and %d boids in avoid radius of %lf.\n",
            i, neighborCount, cuFlockParams.squareNeighborRadius,
            separateCount, cuFlockParams.squareAvoidanceRadius);
    */

    /* Return the resulting composition move */
    return move;
}

__global__ void moveBoids() {
    // WARNING: Method is currently very inefficient, O(n^2) neighbor check
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int boidCount = cuDataParams.boidCount;
    if (i >= boidCount)
        return;

    /* Calculate the combined steers from all forces */
    float2 move = calculate_move(i);
    
    // scale by drive factor
    move *= cuFlockParams.driveFactor;

    // cap at max speed
    if (sqrMagnitude(move) > cuFlockParams.squareMaxSpeed) {
        move = normalize(move);
        move *= cuFlockParams.maxSpeed;
    }

    /* Apply the steers to move position */
    float2 newVelocity = toFloat2(cuDataParams.inData[i].velocity);
    newVelocity += move;
    float2 oldPosition = toFloat2(cuDataParams.inData[i].position);
    float2 newPosition = oldPosition + newVelocity;

    cuDataParams.outData[i].velocity = toVel(newVelocity);
    cuDataParams.outData[i].position = toPos(newPosition);
}

__global__ void kernelPrint() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
   
    if (i >= cuDataParams.boidCount)
        return;

    boid_t boid = cuDataParams.inData[i];
    printf("boid %d indata is %lf %lf %lf %lf\n", i,
            boid.position.x, boid.position.y, boid.velocity.x, boid.velocity.y);
    boid = cuDataParams.outData[i];
    printf("boid %d outdata is %lf %lf %lf %lf\n", i,
            boid.position.x, boid.position.y, boid.velocity.x, boid.velocity.y);

    if (i != 0)
        return;

    printf("Flock Constants:\n");
    printf("driveFactor is %lf\n", cuFlockParams.driveFactor);
    printf("maxSpeed is %lf\n", cuFlockParams.maxSpeed);
    printf("cohesion weight is %lf\n", cuFlockParams.cohesionWeight);
    printf("alignment weight is %lf\n", cuFlockParams.alignmentWeight);
    printf("avoidance weight is %lf\n", cuFlockParams.separationWeight);

    printf("\nData Constants:\n");
    printf("boidCount is %d\n", cuDataParams.boidCount);
    printf("imageWidth is %d\n", cuDataParams.imageWidth);
    printf("imageHeight is %d\n", cuDataParams.imageHeight); 
}

__global__ void kernelPrintPrivate(boid_t *deviceData) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i != 0)
        return;

    printf("Device data:\n");
    printf("first boid at data addr is %lf %lf %lf %lf\n",
            deviceData[0].position.x, deviceData[0].position.y, 
            deviceData[0].velocity.x, deviceData[0].velocity.y);
}

///////////////////////////////////////////////////////////////////////////////////////
CudaBoids::CudaBoids() {
    image = NULL;
    boidCount = 0;
    deviceInData = NULL;
    deviceOutData = NULL;
}

CudaBoids::~CudaBoids() {
    // Free allocated memory
    if (image) {
        delete image;
    }
    
    if (deviceInData) {
        hipFree(deviceInData);
        hipFree(deviceOutData);
    }
}

void CudaBoids::updateScene() {
    dim3 blockDim(256, 1);
    dim3 gridDim((boidCount + blockDim.x - 1) / blockDim.x);

    moveBoids<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
  
    // Now that finished with input data, overwrite with output data for next frame
    copyFrame<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
   
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA Error Occurred: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
    hipDeviceSynchronize();
}

Image *CudaBoids::output() {
    // Need to copy memory from GPU to CPU before returning Image ptr
    hipMemcpy(image->data->boids, deviceOutData, sizeof(boid_t) * boidCount, hipMemcpyDeviceToHost);

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA Error Occurred: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    return image;
}

void CudaBoids::setup(const char *inputName, int num_of_threads) {
    // First, load the input to CPU memory
    FILE *input = fopen(inputName, "r");
    if (!input) {
        printf("Unable to open file: %s.\n", inputName);
        return;
    }

    image = (Image *)malloc(sizeof(Image));

    int dim_x;
    int dim_y;
    fscanf(input, "%d %d\n", &dim_x, &dim_y);
    
    int num_of_boids;
    fscanf(input, "%d\n", &num_of_boids);

    image->width = 2 * dim_x;
    image->height = 2 * dim_y;

    image->data = (group_t*)malloc(sizeof(group_t));
    
    image->data->count = num_of_boids;
    image->data->boids = (boid_t *)calloc(num_of_boids, sizeof(boid_t));

    /* Read the grid dimension and boid information from file */
    
    // Load the coords (x1,y1) for each boid
    int x1, y1;
    for (int i = 0; i < num_of_boids; i++) {
        fscanf(input, "%d %d\n", &x1, &y1);
        image->data->boids[i].position.x = (float)x1;
        image->data->boids[i].position.y = (float)y1;
        image->data->boids[i].velocity.x = (rand() % 3) - 1.f;
        image->data->boids[i].velocity.y = (rand() % 3) - 1.f;
    }

    // Next, setup the cuda device
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaBoids\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // Copy/create necessary data on the GPU
    boidCount = image->data->count;

    hipMalloc(&deviceInData, sizeof(boid_t) * boidCount);
    hipMalloc(&deviceOutData, sizeof(boid_t) * boidCount);
    
    hipMemcpy(deviceInData, image->data->boids, sizeof(boid_t) * boidCount, hipMemcpyHostToDevice);

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA Error Occurred: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    // Initialize parameters in constant memory
    GlobalDataConstants dataParams;
    dataParams.imageWidth = image->width;
    dataParams.imageHeight = image->height;
    dataParams.boidCount = boidCount;
    dataParams.inData = deviceInData;
    dataParams.outData = deviceOutData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuDataParams), &dataParams, sizeof(GlobalDataConstants));

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA Error Occurred: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    GlobalFlockConstants flockParams;
    //flockParams.driveFactor = 10.f;
    flockParams.driveFactor = 10.f;
    //flockParams.maxSpeed = 5.f;
    flockParams.maxSpeed = 5.f;
    flockParams.squareMaxSpeed = flockParams.maxSpeed * flockParams.maxSpeed;
    //flockParams.squareNeighborRadius = 1.5f * 1.5f;
    flockParams.squareNeighborRadius = 75.f * 75.f;
    flockParams.squareAvoidanceRadius = flockParams.squareNeighborRadius * 0.5f * 0.5f;
    flockParams.cohesionWeight = 1.f;
    flockParams.alignmentWeight = 1.f;
    flockParams.separationWeight = 1.f;
    flockParams.centeringWeight = 0.1f;

    hipMemcpyToSymbol(HIP_SYMBOL(cuFlockParams), &flockParams, sizeof(GlobalFlockConstants));
 
    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA Error Occurred: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
    hipDeviceSynchronize();
}
